#include "hip/hip_runtime.h"

//IN THE NAME OF GOD
//creatrd by alireza baneshi
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include <iostream>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#include "gputimer.h"
#include "gpuerrors.h"
#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// TILEX and TILEY is used to set number of threads in a CUDA block 
#define TILE_WIDTH 32
#define TILEY 32
#define TILEX 32



dim3 getDimGrid(const int m, const int n) {
	if(TILEX>TILEY){

dim3 dimGrid(n/(TILEX),n/(TILEX));

	return dimGrid;

}
else{
	dim3 dimGrid(n/(TILEY),n/(TILEY));

	return dimGrid;
}
}

dim3 getDimBlock(const int m, const int n) {
	if(TILEX>TILEY){
dim3 dimBlock(TILEX,TILEX);
	return dimBlock;

}
else{
	dim3 dimBlock(TILEY,TILEY);
	return dimBlock;
}
}



__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {
	 
	if(TILEX==TILEY)						{

    int row = by *TILEY + ty;	 int col = bx * TILEY + tx;
	float tmpVal = 0.0f;
	    __shared__ float bds[TILEX][TILEY];
	    __shared__ float ads[TILEY][TILEX];

    for(int i = 0;i < n / (TILEX);i++){
        ads[ty][tx] = ad[(row * n) + (i * TILEX) + tx];
        bds[ty][tx] = bd[col + (((i * TILEX) + ty) * n)];
        __syncthreads();
for(int k = 0;k < TILEX;k++){
            tmpVal += ads[ty][k] * bds[k][tx];
        }
	__syncthreads();
    }
	// cd[row][col] = ?
	mem2d(cd,m,row,col) = tmpVal;
									}
	else if(TILEY>TILEX)						        {
int row = by *TILEY + ty;	 int col = bx * TILEY + tx;
	float tmpVal = 0.0f;

    __shared__ float bds[TILEX][TILEY];
    __shared__ float ads[TILEY][TILEX];

    for(int i = 0;i < n / (TILEX);i++){
	if(tx<TILEX)
        ads[ty][tx] = ad[(row * n) + (i * TILEX) + tx];
	if(ty<TILEX)
        bds[ty][tx] = bd[col + (((i * TILEX) + ty) * n)];

for(int k = 0;k < TILEX;k++){
        __syncthreads();
            tmpVal += ads[ty][k] * bds[k][tx];
        }
	__syncthreads();
    }
	// cd[row][col] = ?
	mem2d(cd,m,row,col) = tmpVal;



									}
	else if(TILEX>TILEY)						        {
const int tiley = TILEX;
const int tilex = TILEY;

int row = by *tiley + ty;	 int col = bx * tiley + tx;
	float tmpVal = 0.0f;

    __shared__ float bds[tilex][tiley];
    __shared__ float ads[tiley][tilex];

    for(int i = 0;i < n / (tilex);i++){
	if(tx<tilex)
        ads[ty][tx] = ad[(row * n) + (i * tilex) + tx];
	if(ty<tilex)
        bds[ty][tx] = bd[col + (((i * tilex) + ty) * n)];

for(int k = 0;k < tilex;k++){
        __syncthreads();
            tmpVal += ads[ty][k] * bds[k][tx];
        }
	__syncthreads();
    }
	// cd[row][col] = ?
	mem2d(cd,m,row,col) = tmpVal;



									}



}






//-----------------------------------------------------------------------------
void gpuKernel(const float* const  a, const float* const b, float* c, const int m, const int n) {
	
	
	float* ad;
        float* bd;
        float* cd;
        float* ad1;
        float* bd1;
	float r =n/2;
        float r1 =m-1;
        float* cd2;
        bd1 = (float*)malloc(n*(n/4) * sizeof(float));
	ad1 = (float*)malloc(n*(n/4) * sizeof(float));
	cd2 = (float*)malloc(n*(n/4) * sizeof(float));
	
if(m<14){
   dim3 dimGrid = getDimGrid(m,n); 
	dim3 dimBlock = getDimBlock(m,n); 




    HANDLE_ERROR(hipMalloc((void**)&ad, n*(n) * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&bd, n*(n) * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&cd, n*(n) * sizeof(float)));

HANDLE_ERROR(hipMemcpy(ad, a, n*(n) * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(bd, b, n*(n) * sizeof(float), hipMemcpyHostToDevice));
    kernelFunc<<< dimGrid,dimBlock >>>(ad, bd, cd, m, n);
    HANDLE_ERROR(hipMemcpy(c, cd, n*(n) * sizeof(float), hipMemcpyDeviceToHost));
        


                HANDLE_ERROR(hipFree(ad));
		HANDLE_ERROR(hipFree(bd));
                HANDLE_ERROR(hipFree(cd));



}
else
{


	for (int i = 0 ; i< n/2 ;i++){

		for (int k = 0; k<(n/2) ;k++){

			bd1[k+(i*n/2)] = b[k+(i*n)];
			ad1[k+(i*n/2)] = a[k+(i*n)];
			}

	      }
    HANDLE_ERROR(hipMalloc((void**)&ad, n*(n/4) * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&bd, n*(n/4) * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&cd, n*(n/4) * sizeof(float)));


     
    HANDLE_ERROR(hipMemcpy(ad, ad1, n*(n/4) * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(bd, bd1, n*(n/4) * sizeof(float), hipMemcpyHostToDevice));
dim3 dimGrid = getDimGrid(r1,r); 
	dim3 dimBlock = getDimBlock(r1,r); 
    kernelFunc<<< dimGrid,dimBlock >>>(ad, bd, cd, r1, r);
//std::cout<<1;
    HANDLE_ERROR(hipMemcpy(cd2, cd, n*(n/4) * sizeof(float), hipMemcpyDeviceToHost));
        for (int i = 0 ; i< n/2 ;i++)
		for (int k = 0; k<(n/2) ;k++)
			c[k+n*i] = cd2[k+(n*i/2)];
/////////////////////////////////////////////////////////////////////////////////////////////////////////


for (int i = 0 ; i< n/2 ;i++){

		for (int k = 0; k<(n/2) ;k++){
			bd1[k+(i*n/2)] = b[k+(i*n)+(n*n/2)];
			ad1[k+(i*n/2)] = a[k+(i*n)+(n/2)];
			}

	      }


    HANDLE_ERROR(hipMemcpy(ad, ad1, n*(n/4) * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(bd, bd1, n*(n/4) * sizeof(float), hipMemcpyHostToDevice));
    kernelFunc<<< dimGrid,dimBlock >>>(ad, bd, cd, r1, r);
    HANDLE_ERROR(hipMemcpy(cd2, cd, n*(n/4) * sizeof(float), hipMemcpyDeviceToHost));
         for (int i = 0 ; i< n/2 ;i++)
		for (int k = 0; k<(n/2) ;k++)
			c[k+n*i] += cd2[k+(n*i/2)];
/////////////////////////////////////////////////////////////////////////////////////////////////////////

/////////////////////////////////////////////////////////////////////////////////////////////////////////


for (int i = 0 ; i< n/2 ;i++){

		for (int k = 0; k<(n/2) ;k++){

			bd1[k+(i*n/2)] = b[k+(i*n)+(n/2)+(n*n/2)];
			}

	      }


    HANDLE_ERROR(hipMemcpy(bd, bd1, n*(n/4) * sizeof(float), hipMemcpyHostToDevice));
    kernelFunc<<< dimGrid,dimBlock >>>(ad, bd, cd, r1, r);
    HANDLE_ERROR(hipMemcpy(cd2, cd, n*(n/4) * sizeof(float), hipMemcpyDeviceToHost));
         for (int i = 0 ; i< n/2 ;i++)
		for (int k = 0; k<(n/2) ;k++)
			c[k+n*i+n/2] = cd2[k+(n*i/2)];
/////////////////////////////////////////////////////////////////////////////////////////////////////////



for (int i = 0 ; i< n/2 ;i++){

		for (int k = 0; k<(n/2) ;k++){


			bd1[k+(i*n/2)] = b[k+(i*n)+(n/2)];
			ad1[k+(i*n/2)] = a[k+(i*n)];
			}

	      }



    HANDLE_ERROR(hipMemcpy(ad, ad1, n*(n/4) * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(bd, bd1, n*(n/4) * sizeof(float), hipMemcpyHostToDevice));
    kernelFunc<<< dimGrid,dimBlock >>>(ad, bd, cd, r1, r);
    HANDLE_ERROR(hipMemcpy(cd2, cd, n*(n/4) * sizeof(float), hipMemcpyDeviceToHost));
         for (int i = 0 ; i< n/2 ;i++)
		for (int k = 0; k<(n/2) ;k++)
			c[k+n*i+n/2] += cd2[k+(n*i/2)];
/////////////////////////////////////////////////////////////////////////////////////////////////////////

/////////////////////////////////////////////////////////////////////////////////////////////////////////


for (int i = 0 ; i< n/2 ;i++){

		for (int k = 0; k<(n/2) ;k++){

			
			ad1[k+(i*n/2)] = a[k+(i*n)+(n*n/2)];

		}

	      }


    HANDLE_ERROR(hipMemcpy(ad, ad1, n*(n/4) * sizeof(float), hipMemcpyHostToDevice));
    kernelFunc<<< dimGrid,dimBlock >>>(ad, bd, cd, r1, r);
    HANDLE_ERROR(hipMemcpy(cd2, cd, n*(n/4) * sizeof(float), hipMemcpyDeviceToHost));
         for (int i = 0 ; i< n/2 ;i++)
		for (int k = 0; k<(n/2) ;k++)
			c[k+n*i+n/2+(n*n/2)] = cd2[k+(n*i/2)];
/////////////////////////////////////////////////////////////////////////////////////////////////////////


for (int i = 0 ; i< n/2 ;i++){

		for (int k = 0; k<(n/2) ;k++){

			bd1[k+(i*n/2)] = b[k+(i*n)+(n/2)+(n*n/2)];

			ad1[k+(i*n/2)] = a[k+(i*n)+(n/2)+(n*n/2)];

		}

	      }



    HANDLE_ERROR(hipMemcpy(ad, ad1, n*(n/4) * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(bd, bd1, n*(n/4) * sizeof(float), hipMemcpyHostToDevice));
    kernelFunc<<< dimGrid,dimBlock >>>(ad, bd, cd, r1, r);
    HANDLE_ERROR(hipMemcpy(cd2, cd, n*(n/4) * sizeof(float), hipMemcpyDeviceToHost));
         for (int i = 0 ; i< n/2 ;i++)
		for (int k = 0; k<(n/2) ;k++)
			c[k+n*i+n/2+(n*n/2)] += cd2[k+(n*i/2)];
/////////////////////////////////////////////////////////////////////////////////////////////////////////

/////////////////////////////////////////////////////////////////////////////////////////////////////////



for (int i = 0 ; i< n/2 ;i++){

		for (int k = 0; k<(n/2) ;k++){

			bd1[k+(i*n/2)] = b[k+(i*n)+(n*n/2)];
			}

	      }



    HANDLE_ERROR(hipMemcpy(bd, bd1, n*(n/4) * sizeof(float), hipMemcpyHostToDevice));
    kernelFunc<<< dimGrid,dimBlock >>>(ad, bd, cd, r1, r);
    HANDLE_ERROR(hipMemcpy(cd2, cd, n*(n/4) * sizeof(float), hipMemcpyDeviceToHost));
         for (int i = 0 ; i< n/2 ;i++)
		for (int k = 0; k<(n/2) ;k++)
			c[k+n*i+(n*n/2)] = cd2[k+(n*i/2)];
/////////////////////////////////////////////////////////////////////////////////////////////////////////



for (int i = 0 ; i< n/2 ;i++){

		for (int k = 0; k<(n/2) ;k++){

			bd1[k+(i*n/2)] = b[k+(i*n)];			
			ad1[k+(i*n/2)] = a[k+(i*n)+(n*n/2)];


		}

	      }


    HANDLE_ERROR(hipMemcpy(ad, ad1, n*(n/4) * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(bd, bd1, n*(n/4) * sizeof(float), hipMemcpyHostToDevice));
    kernelFunc<<< dimGrid,dimBlock >>>(ad, bd, cd, r1, r);
    HANDLE_ERROR(hipMemcpy(cd2, cd, n*(n/4) * sizeof(float), hipMemcpyDeviceToHost));
         for (int i = 0 ; i< n/2 ;i++)
		for (int k = 0; k<(n/2) ;k++)
			c[k+n*i+(n*n/2)] += cd2[k+(n*i/2)];



                HANDLE_ERROR(hipFree(ad));
		HANDLE_ERROR(hipFree(bd));
                HANDLE_ERROR(hipFree(cd));

	//GpuTimer timer;

    //timer.Start();*/
}
}

